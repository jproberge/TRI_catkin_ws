#include "hip/hip_runtime.h"
/*
 * This is part of the Bayesian Object Tracking (bot),
 * (https://github.com/bayesian-object-tracking)
 *
 * Copyright (c) 2015 Max Planck Society,
 * 				 Autonomous Motion Department,
 * 			     Institute for Intelligent Systems
 *
 * This Source Code Form is subject to the terms of the GNU General Public
 * License License (GNU GPL). A copy of the license can be found in the LICENSE
 * file distributed with this source code.
 */

/*
 * This file implements a part of the algorithm published in:
 *
 * M. Wuthrich, P. Pastor, M. Kalakrishnan, J. Bohg, and S. Schaal.
 * Probabilistic Object Tracking using a Range Camera
 * IEEE Intl Conf on Intelligent Robots and Systems, 2013
 * http://arxiv.org/abs/1505.00241
 *
 */

/**
 * \file cuda_likelihood_evaluator.cu
 * \author Claudia Pfreundt (claudilein@gmail.com)
 * \date November 2015
 */

#define DEBUG

#define VECTOR_DIM 3
#define MATRIX_DIM 9

#include <GL/glut.h>
#include <fl/util/profiling.hpp>
#include <dbot/gpu/cuda_likelihood_evaluator.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>


#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <math.h>
#include <hip/hip_math_constants.h>


using namespace std;

// ====================== CUDA CONSTANT VALUES ======================= //


// used in propagateOcclusion
__constant__ float g_p_occluded_occluded;
__constant__ float g_one_div_c_minus_one;
__constant__ float g_log_c;


// used in prob
__constant__ float g_one_minus_tail_weight;
__constant__ float g_model_sigma;
__constant__ float g_sigma_factor;
__constant__ float g_tail_weight_div_max_depth;
__constant__ float g_exponential_rate;
__constant__ float g_one_div_sqrt_of_two;
__constant__ float g_one_div_sqrt_of_two_pi;


// used in compare
__constant__ float g_initial_occlusion_prob;

// texture for OpenGL interop
texture<float, hipTextureType2D, hipReadModeElementType> texture_reference;





// ************************************************************************************** //
// ************************************************************************************** //
// ================================== CUDA KERNELS ====================================== //
// ************************************************************************************** //
// ************************************************************************************** //

// ============================================================================================= //
// ====================== DEVICE kernels - to be called by other kernels ======================= //
// ============================================================================================= //


// ======================= helper functions for compare (observation model)  ======================= //


__device__ float propagate_occlusion(float initial_p_source, float time) {
    if (isnan(time)) {
        return initial_p_source;
    }
    float pow_c_time = __expf(time * g_log_c);
    return 1 - (pow_c_time * (1 - initial_p_source) + (1. - g_p_occluded_occluded) * (pow_c_time - 1.) * g_one_div_c_minus_one);
}



__device__ float prob(float observation, float prediction, bool occluded)
{
    // todo: if the prediction is infinite, the prob should not depend on occlusion. it does not matter
    // for the algorithm right now, but it should be changed

    float sigma = g_model_sigma + g_sigma_factor * observation * observation;
    float sigma_sq = sigma * sigma;

    if(!occluded)
    {
        if(isinf(prediction)) // if the prediction is infinite we return the limit
            return g_tail_weight_div_max_depth;
        else {
            float pred_minus_obs = prediction - observation;
            return g_tail_weight_div_max_depth
                    + __fdividef(g_one_minus_tail_weight * __expf(- __fdividef(pred_minus_obs * pred_minus_obs, (2 * sigma_sq)))
                    * g_one_div_sqrt_of_two_pi, sigma);
        }
    }
    else
    {
        if(isinf(prediction)) // if the prediction is infinite we return the limit
            return g_tail_weight_div_max_depth +
                    g_one_minus_tail_weight * g_exponential_rate *
                    __expf(0.5 * g_exponential_rate * (-2 * observation + g_exponential_rate * sigma_sq));

        else
            return g_tail_weight_div_max_depth +
                    g_one_minus_tail_weight * g_exponential_rate *
                    __expf(0.5 * g_exponential_rate * (2 * (prediction - observation) + g_exponential_rate * sigma_sq))
                    * __fdividef((1 + erff(__fdividef((prediction - observation + g_exponential_rate * sigma_sq) * g_one_div_sqrt_of_two, sigma))),
                    (2 * (__expf(prediction * g_exponential_rate) - 1)));
    }
}




// ============================================================================================= //
// ========================= GLOBAL kernels - to be called by CPU code ========================= //
// ============================================================================================= //



__global__ void evaluate_kernel(float *observations, float* old_occlusion_probs, float* new_occlusion_probs, int* occlusion_image_indices, int nr_pixels,
                                 float *d_log_likelihoods, float delta_time, int n_poses, int n_rows, int n_cols, bool update_occlusions) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    if (block_id < n_poses) {

        int pixel_nr = threadIdx.x;

        // OpenGL contructs the texture so that the left lower edge is (0,0), but our observations texture
        // has its (0,0) in the upper left corner, so we need to reverse the reads from the OpenGL texture.
        float texture_array_index_x = blockIdx.x * n_cols + pixel_nr % n_cols;

        float texture_array_index_y = gridDim.y * n_rows - 1 - (blockIdx.y * n_rows + __fdividef(pixel_nr, n_cols));

        float depth;
        float observed_depth;
        float occlusion_prob = g_initial_occlusion_prob;
        float local_sum_of_likelihoods = 0;
        float p_obsIpred_vis, p_obsIpred_occl, p_obsIinf;

        __shared__ float log_likelihoods;
        __shared__ int occlusion_image_index;

        if (threadIdx.x == 0) {
            log_likelihoods = 0;
            occlusion_image_index = occlusion_image_indices[block_id];
        }

        __syncthreads();

        float* occlusion_probs = old_occlusion_probs;
        int occlusion_pixel_index= occlusion_image_index * nr_pixels + pixel_nr;

        if (update_occlusions) {
            // copy occlusion probabilities from the old particles
            int index_from_occlusion = occlusion_image_indices[block_id] * nr_pixels;
            int index_to_occlusion = block_id * nr_pixels;

            while (pixel_nr < nr_pixels) {
                new_occlusion_probs[index_to_occlusion + pixel_nr] = old_occlusion_probs[index_from_occlusion + pixel_nr];
                pixel_nr += blockDim.x;
            }

            // change occlusion prob array to the new one and change the global index
            occlusion_probs = new_occlusion_probs;
            // reset pixel_nr for following loop
            pixel_nr = threadIdx.x;

            occlusion_pixel_index= block_id * nr_pixels + pixel_nr;
        }


        while (pixel_nr < nr_pixels ) {

            depth = tex2D(texture_reference, texture_array_index_x, texture_array_index_y);
            observed_depth = observations[pixel_nr];

            occlusion_prob = propagate_occlusion(occlusion_probs[occlusion_pixel_index], delta_time);
            if (update_occlusions) occlusion_probs[occlusion_pixel_index] = occlusion_prob;


            if (depth != 0 && !isnan(observed_depth)) {

                // prob of observation given prediction, knowing that the object is not occluded
                p_obsIpred_vis = prob(observed_depth, depth, false) * (1 - occlusion_prob);
                // prob of observation given prediction, knowing that the object is occluded
                p_obsIpred_occl = prob(observed_depth, depth, true) * occlusion_prob;
                // prob of observation given no intersection
                p_obsIinf = prob(observed_depth, HIP_INF_F, true);

                local_sum_of_likelihoods += __logf(__fdividef((p_obsIpred_vis + p_obsIpred_occl), p_obsIinf));


                if(update_occlusions) {
                    // we update the occlusion probability with the observations
                    occlusion_probs[occlusion_pixel_index] = 1 - __fdividef(p_obsIpred_vis, (p_obsIpred_vis + p_obsIpred_occl));
                }
            }

            pixel_nr += blockDim.x;
            occlusion_pixel_index += blockDim.x;
            texture_array_index_x = blockIdx.x * n_cols + pixel_nr % n_cols;
            texture_array_index_y = gridDim.y * n_rows - (blockIdx.y * n_rows + (pixel_nr / n_cols) + 1);
        }

        atomicAdd(&log_likelihoods, local_sum_of_likelihoods);

        __syncthreads();

        if (threadIdx.x == 0) {
            d_log_likelihoods[block_id] = log_likelihoods;
        }
    } else {
        __syncthreads();
    }

}






// ************************************************************************************** //
// ************************************************************************************** //
// ========================== cuda_likelihood_evaluator MEMBER FUNCTIONS ============================== //
// ************************************************************************************** //
// ************************************************************************************** //


CudaEvaluator::CudaEvaluator(const int nr_rows,
                       const int nr_cols) :

    nr_rows_(nr_rows),
    nr_cols_(nr_cols)
{

    hipDeviceProp_t  props;
    int device_number;

    memset( &props, 0, sizeof( hipDeviceProp_t ) );
    props.major = 2;
    props.minor = 0;
    hipChooseDevice( &device_number, &props );
    #ifdef DEBUG
        check_cuda_error("No device with compute capability > 2.0 found");
    #endif

    /* tell CUDA which device we will be using for graphic interop.
     * Requires that the CUDA device be specified by
     * cudaGLSetGLDevice() before any other runtime calls. */

//    cudaGLSetGLDevice( device_number );
    #ifdef DEBUG
        check_cuda_error("cudaGLsetGLDevice");
    #endif

    hipGetDeviceProperties(&props, device_number);     // we will run the program only on one graphics card, the first one we can find = 0

    cuda_device_properties_ = props;

    #ifdef DEBUG
        cout << "Your device has the following properties: " << endl
             << "CUDA Version: " << props.major << "." << props.minor << endl
             << "Number of multiprocessors: " << props.multiProcessorCount << endl;
    #endif

    /* each multiprocessor has various KB of memory (64 KB for the GTX 560 Ti 448) which can be subdivided
     * into L1 cache or shared memory. If you don't need a lot of shared memory set this to prefer L1. */
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);


    d_occlusion_probs_ = NULL;
    d_occlusion_probs_copy_ = NULL;
    d_observations_ = NULL;
    d_log_likelihoods_ = NULL;
    d_occlusion_indices_ = NULL;

    set_nr_threads(DEFAULT_NR_THREADS);
}



void CudaEvaluator::init(const float initial_occlusion_prob, const float p_occluded_occluded, const float p_occluded_visible,
                      const float tail_weight, const float model_sigma, const float sigma_factor, const float max_depth, const float exponential_rate) {

    occlusion_time_ = 0;
    occlusion_prob_default_ = initial_occlusion_prob;

    // precompute constants that are used in high-performance kernels later
    float c = p_occluded_occluded - p_occluded_visible;
    float tail_weight_div_max_depth = tail_weight / max_depth;
    float one_minus_tail_weight = 1.0f - tail_weight;
    float one_div_c_minus_one = 1.0f / (c - 1.0f);
    float one_div_sqrt_of_two = 1.0f / sqrt(2);
    float one_div_sqrt_of_two_pi = 1.0f / sqrt(2 * M_PI);
    float log_c = log(c);



    // copy constants to GPU memory
    hipMemcpyToSymbol(HIP_SYMBOL(g_initial_occlusion_prob), &initial_occlusion_prob, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol initial_occlusion_prob -> g_initial_occlusion_prob");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_one_div_c_minus_one), &one_div_c_minus_one, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol one_div_c_minus_one -> g_one_div_c_minus_one");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_one_div_sqrt_of_two), &one_div_sqrt_of_two, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol one_div_sqrt_of_two -> g_one_div_sqrt_of_two");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_one_div_sqrt_of_two_pi), &one_div_sqrt_of_two_pi, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol one_div_sqrt_of_two_pi -> g_one_div_sqrt_of_two_pi");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_log_c), &log_c, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol log_c -> g_log_c");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_p_occluded_occluded), &p_occluded_occluded, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol p_occluded_occluded -> g_p_occluded_occluded");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_one_minus_tail_weight), &one_minus_tail_weight, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol one_minus_tail_weight -> g_one_minus_tail_weight");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_model_sigma), &model_sigma, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol model_sigma -> g_model_sigma");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_sigma_factor), &sigma_factor, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol sigma_factor -> g_sigma_factor");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_tail_weight_div_max_depth), &tail_weight_div_max_depth, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol tail_weight_div_max_depth -> g_tail_weight_div_max_depth");
    #endif

    hipMemcpyToSymbol(HIP_SYMBOL(g_exponential_rate), &exponential_rate, sizeof(float), 0, hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpyToSymbol exponential_rate -> g_exponential_rate");
    #endif


    constants_initialized_ = true;
}





void CudaEvaluator::weigh_poses(const bool update_occlusions, vector<float> &log_likelihoods) {
    if (observations_set_ && occlusion_indices_set_
            && memory_allocated_ && number_of_poses_set_ && constants_initialized_
            && texture_array_mapped_) {

        double delta_time = observation_time_ - occlusion_time_;
        if(update_occlusions) occlusion_time_ = observation_time_;


        evaluate_kernel <<< grid_dimension_, nr_threads_ >>> (d_observations_, d_occlusion_probs_, d_occlusion_probs_copy_, d_occlusion_indices_, nr_cols_ * nr_rows_,
                                               d_log_likelihoods_, delta_time, nr_poses_, nr_rows_, nr_cols_, update_occlusions);
        #ifdef DEBUG
            check_cuda_error("compare kernel call");
        #endif

        hipDeviceSynchronize();
        #ifdef DEBUG
            check_cuda_error("hipDeviceSynchronize compare_multiple");
        #endif

        // switch to new / copied occlusion probabilities
        if (update_occlusions) {
            float *tmp_pointer;
            tmp_pointer = d_occlusion_probs_;
            d_occlusion_probs_ = d_occlusion_probs_copy_;
            d_occlusion_probs_copy_ = tmp_pointer;
        }


        hipMemcpy(&log_likelihoods[0], d_log_likelihoods_, nr_poses_ * sizeof(float), hipMemcpyDeviceToHost);
        #ifdef DEBUG
            check_cuda_error("hipMemcpy d_log_likelihoods -> log_likelihoods");
        #endif

        hipDeviceSynchronize();
        #ifdef DEBUG
            check_cuda_error("hipDeviceSynchronize compare");
        #endif
    } else {
        std::cout << "WARNING (CUDA): It seems you forgot to do one of the following: set observation image, set occlusion"
                  << " indices, set number of poses, allocate memory, map texture to texture array or inisitialize constants." << std::endl;
    }

}







// ===================================================================================== //
// =============================== CUDA EVALUATOR SETTERS ================================= //
// ===================================================================================== //

void CudaEvaluator::set_nr_threads(const int nr_threads) {
    if (nr_threads > cuda_device_properties_.maxThreadsDim[0]) {
        std::cout << "ERROR (CUDA): The number of threads you requested ("
                  << nr_threads << ") is not supported my CUDA. Max #threads: "
                  << cuda_device_properties_.maxThreadsDim[0] << std::endl;
        exit(-1);
    }

    nr_threads_ = nr_threads;
}



void CudaEvaluator::set_observations(const float* observations, const float observation_time) {

    if (nr_rows_ * nr_cols_ > observations_size_) {
        std::cout << "ERROR (CUDA) in set_observations: You exceeded "
                  << "(" << nr_rows_ * nr_cols_ << ")"
                  << "the memory space that was allocated for the observation "
                  << "values (" << observations_size_ << ")." << std::endl;
        exit(-1);
    }

    observation_time_ = observation_time;

    hipMemcpy(d_observations_, observations, nr_cols_ * nr_rows_ * sizeof(float), hipMemcpyHostToDevice);
    #ifdef DEBUG
        check_cuda_error("hipMemcpy observations -> d_observations_");
    #endif
    hipDeviceSynchronize();
    #ifdef DEBUG
        check_cuda_error("hipDeviceSynchronize set_observations");
    #endif

    observations_set_ = true;
}



void CudaEvaluator::set_occlusion_indices(const int* occlusion_indices,
                                          const int array_size) {

    if (array_size > max_nr_poses_) {
        std::cout << "ERROR (CUDA): The amount of occlusion indices sent to "
                  << "the GPU (" << array_size << ") exceeds the amount of "
                  << "maximum poses (" << max_nr_poses_ << ")." << std::endl;
        exit(-1);
    }

    hipMemcpy(d_occlusion_indices_, occlusion_indices,
               array_size * sizeof(int), hipMemcpyHostToDevice);

    #ifdef DEBUG
        check_cuda_error("hipMemcpy occlusion_indices -> d_occlusion_indices");
    #endif
    hipDeviceSynchronize();
    #ifdef DEBUG
        check_cuda_error("hipDeviceSynchronize set_occlusion_indices");
    #endif

    occlusion_indices_set_ = true;
}


void CudaEvaluator::set_resolution(const int nr_rows, const int nr_cols) {

    nr_rows_ = nr_rows;
    nr_cols_ = nr_cols;

}


void CudaEvaluator::set_occlusion_probabilities(const float* occlusion_probabilities,
                                                const int array_size) {

    if (array_size > occlusion_probs_size_) {
        std::cout << "ERROR (CUDA) in set_occlusion_probabilities: You exceeded "
                  << "(" << array_size << ")"
                  << "the memory space that was allocated for the occlusion "
                  << "probabilities (" << occlusion_probs_size_ << "." << std::endl;
        exit(-1);
    }

    hipMemcpy(d_occlusion_probs_, occlusion_probabilities,
               array_size * sizeof(float), hipMemcpyHostToDevice);

    #ifdef DEBUG
        check_cuda_error("hipMemcpy occlusion_probabilities -> d_occlusion_probs_");
    #endif
    hipDeviceSynchronize();
    #ifdef DEBUG
        check_cuda_error("hipDeviceSynchronize set_occlusion_probabilities");
    #endif
}


void CudaEvaluator::map_texture_to_texture_array(const hipArray_t texture_array) {

    d_texture_array_ = texture_array;
    hipBindTextureToArray(texture_reference, d_texture_array_);

    #ifdef DEBUG
        check_cuda_error("hipBindTextureToArray");
    #endif

    texture_array_mapped_ = true;
}


void CudaEvaluator::allocate_memory_for_max_poses(int nr_poses,
                                                  int nr_poses_per_row,
                                                  int nr_poses_per_col) {
    if (constants_initialized_) {

        // check limitation by global memory size
        int constant_need, per_pose_need;
        get_memory_need_parameters(nr_rows_, nr_cols_,
                                   constant_need, per_pose_need);
        int memory_needs = constant_need + nr_poses * per_pose_need;

        if (memory_needs > cuda_device_properties_.totalGlobalMem) {
            std::cout << "ERROR (CUDA): Not enough memory to allocate " << nr_poses
                      << " poses." << std::endl;
            exit(-1);
        }

        // check limitation by texture and grid size
        if (nr_poses_per_row * nr_cols_ > cuda_device_properties_.maxTexture2D[0] ||
            nr_poses_per_col * nr_rows_ > cuda_device_properties_.maxTexture2D[1] ||
            nr_poses_per_row > cuda_device_properties_.maxGridSize[0] ||
            nr_poses_per_col > cuda_device_properties_.maxGridSize[1]) {
            std::cout << "ERROR (CUDA): Exceeding maximum texture or grid size with"
                      << nr_poses_per_row << " x " << nr_poses_per_col << " poses"
                      << " at resolution " << nr_rows_ << " x " << nr_cols_ << std::endl;
            exit(-1);
        }

        max_nr_poses_ = nr_poses;
        max_nr_poses_per_row_ = nr_poses_per_row;
        max_nr_poses_per_column_ = nr_poses_per_col;

        grid_dimension_ = dim3(nr_poses_per_row, nr_poses_per_col);


        // reallocate arrays
        allocate(d_log_likelihoods_, sizeof(float) * max_nr_poses_);
        allocate(d_occlusion_indices_, sizeof(int) * max_nr_poses_);
        occlusion_probs_size_ = nr_rows_ * nr_cols_ * max_nr_poses_;
        allocate(d_occlusion_probs_, occlusion_probs_size_ * sizeof(float));
        allocate(d_occlusion_probs_copy_, occlusion_probs_size_ * sizeof(float));
        observations_size_ = nr_rows_ * nr_cols_;
        allocate(d_observations_, observations_size_ * sizeof(float));

        vector<float> initial_occlusion_probs (nr_rows_ * nr_cols_ * max_nr_poses_,
                                               occlusion_prob_default_);

        hipMemcpy(d_occlusion_probs_, &initial_occlusion_probs[0], occlusion_probs_size_ * sizeof(float), hipMemcpyHostToDevice);
        #ifdef DEBUG
            check_cuda_error("hipMemcpy occlusion_prob_default_ -> d_occlusion_probs_");
        #endif

        // initialize log likelihoods with 0
        hipMemset(d_log_likelihoods_, 0, sizeof(float) * max_nr_poses_);
        #ifdef DEBUG
            check_cuda_error("hipMemset d_log_likelihoods");
        #endif

        hipDeviceSynchronize();
        #ifdef DEBUG
            check_cuda_error("hipDeviceSynchronize allocate_memory_for_max_poses");
        #endif

        memory_allocated_ = true;
    } else {
        std::cout << "WARNING (CUDA): It seems you forgot to call init() to "
                  << "initialize the constants before calling "
                  << "allocate_memory_for_max_poses" << std::endl;
    }
}


void CudaEvaluator::set_number_of_poses(int nr_poses) {
    if (memory_allocated_) {
        if (nr_poses > max_nr_poses_) {
            std::cout << "ERROR (CUDA): You tried to evaluate more poses ("
                      << nr_poses << ") than specified by max_poses ("
                      << max_nr_poses_ << ")." << std::endl;
            exit(-1);
        }

        nr_poses_ = nr_poses;
        int nr_poses_per_row = min(max_nr_poses_per_row_, nr_poses);
        int nr_poses_per_column = min(max_nr_poses_per_column_,
                                   (int) ceil(nr_poses / (float) nr_poses_per_row));

        grid_dimension_ = dim3(nr_poses_per_row, nr_poses_per_column);

        number_of_poses_set_ = true;
    } else {
        std::cout << "WARNING (CUDA): It seems you forgot to call "
                  << "allocate_memory_for_max_poses before calling "
                  << "set_number_of_poses." << std::endl;
    }
}






// ===================================================================================== //
// =============================== CUDA EVALUATOR GETTERS ================================= //
// ===================================================================================== //


int CudaEvaluator::get_max_nr_threads() {
    return cuda_device_properties_.maxThreadsDim[0];
}

int CudaEvaluator::get_default_nr_threads() {
    return DEFAULT_NR_THREADS;
}

int CudaEvaluator::get_warp_size() {
    return cuda_device_properties_.warpSize;
}

hipDeviceProp_t CudaEvaluator::get_device_properties() {
    return cuda_device_properties_;
}


void CudaEvaluator::get_memory_need_parameters(int nr_rows, int nr_cols,
                                int& constant_need, int& per_pose_need) {
    constant_need = nr_rows * nr_cols * sizeof(float);
    per_pose_need = (3 + 2 * nr_rows * nr_cols) * sizeof(float);
}

vector<float> CudaEvaluator::get_occlusion_probabilities(int state_id) {
    if (memory_allocated_) {
        float* occlusion_probabilities = (float*) malloc(nr_rows_ * nr_cols_ * sizeof(float));
        int offset = state_id * nr_rows_ * nr_cols_;
        hipMemcpy(occlusion_probabilities, d_occlusion_probs_ + offset, nr_rows_ * nr_cols_ * sizeof(float), hipMemcpyDeviceToHost);

        #ifdef DEBUG
            check_cuda_error("hipMemcpy d_occlusion_probabilities -> occlusion_probabilities");
        #endif

        vector<float> occlusion_probabilities_vector;
        for (int i = 0; i < nr_rows_ * nr_cols_; i++) {
            occlusion_probabilities_vector.push_back(occlusion_probabilities[i]);
        }
        free(occlusion_probabilities);
        return occlusion_probabilities_vector;
    } else {
        std::cout << "WARNING (CUDA): It seems you forgot to call "
                  << "allocate_memory_for_max_poses before calling "
                  << "get_occlusion_probabilities." << std::endl;
        vector<float> dummy;
        return dummy;
    }
}




// ===================================================================================== //
// ========================== CUDA EVALUATOR HELPER FUNCTIONS ============================= //
// ===================================================================================== //




template <typename T> void CudaEvaluator::allocate(T * &pointer, size_t size) {
    hipFree(pointer);
    hipMalloc((void **) &pointer, size);
#ifdef DEBUG
    check_cuda_error("hipMalloc failed");
#endif
}



void CudaEvaluator::check_cuda_error(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

// ===================================================================================== //
// ============================ CUDA EVALUATOR DESTRUCTOR  ================================ //
// ===================================================================================== //




CudaEvaluator::~CudaEvaluator() {
    hipFree(d_occlusion_probs_);
    hipFree(d_occlusion_probs_copy_);
    hipFree(d_observations_);
    hipFree(d_log_likelihoods_);
    hipFree(d_occlusion_indices_);
    hipDeviceReset();
}

